//written by miaojingshi 201708 
#include <vector>

#include "caffe/layers/euclidean_loss_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
// LOG(INFO) << "estedmated count" << bottom[0]->cpu_data()[0];
 // LOG(INFO) << "groundtruth count" << bottom[1]->cpu_data()[0];
 caffe_gpu_set(count, Dtype(1), diffdiv_.mutable_gpu_data()); 
 caffe_gpu_axpy(count, Dtype(1), bottom[1]->gpu_data(), diffdiv_.mutable_gpu_data());
   
 caffe_gpu_div(count, diff_.gpu_data(), diffdiv_.gpu_data(), diff_.mutable_gpu_data());
 caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
 //dot = diff_.asum_data();   
 Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void EuclideanLossNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
          caffe_gpu_div(bottom[i]->count(), bottom[i]->gpu_data(), diffdiv_.gpu_data(), bottom[i]->mutable_gpu_data());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossNormLayer);

}  // namespace caffe
